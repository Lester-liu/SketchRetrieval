#include "hip/hip_runtime.h"
//
// Created by lyx on 17/11/15.
//

#include "kernel.h"
#include <>

#define BLOCK_SIZE 256

template <typename T>
__device__ T square(const T x) {
    return x * x;
}

template <typename T>
__global__ void kernel_sequence(T *d_m, int n, T start, T step) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        d_m[i] = start + step * i;
}

template <typename T>
__global__ void kernal_set_value(T *d_m, int n, T value) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        d_m[i] = value;
}

template <typename T>
__global__ void kernel_scale(T *d_m, float *d_scale, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        d_m[i] *= 1 + d_scale[i];
}

template <typename T>
__global__ void kernel_square_minus(T *d_center, int n, int m, T *point, T *d_diff) {
    int id_center = blockIdx.x; // blockDim.x == dim && max(blockIdx.x) == center_count
    int id_pos = threadIdx.x; // max(threadIdx.x) == dim
    if (id_pos < n && id_center < m)
        d_diff[id_center * blockDim.x + id_pos] = square(d_center[id_center * blockDim.x + id_pos] - point[id_pos]);
}

template <typename T>
__global__ void kernel_transpose_scale(T *d_center, int n, int m, T *d_center_tmp, T *size) {
    // each column of d_center (n * m) correspond to each row of d_center_tmp * size[row]
    int row = threadIdx.x;
    int col = blockIdx.x;
    // m == center_count && n == dim
    if (row + m * col < n * m && row * n + col < n * m) {
        if (size[col] != 0)
            d_center[row + m * col] = d_center_tmp[row * n + col] / size[col];
        else
            d_center[row + m * col] = d_center_tmp[row * n + col];
    }
}

template <typename T>
__global__ void kernel_uniform_scale(T *x, int n, T min, T max) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        x[i] = x[i] * (max - min) + min;
}

void set_value(float *d_m, int n, float value) {
    if (value == 0)
        callCuda(hipMemset(d_m, 0, sizeof(float) * n));
    else
        kernal_set_value<<<(n - 1 + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(d_m, n, value);
}

void set_value(int *d_m, int n, int value) {
    if (value == 0)
        callCuda(hipMemset(d_m, 0, sizeof(int) * n));
    else
        kernal_set_value<<<(n - 1 + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(d_m, n, value);
}

void set_sequence(int *d_m, int n, int start, int step) {
    kernel_sequence<<<(n - 1 + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(d_m, n, start, step);
}

void transpose_scale(float *d_center, int n, int m, float *d_center_transpose, float *d_cluster_size) {
    kernel_transpose_scale<<<m, n>>>(d_center, m, n, d_center_transpose, d_cluster_size);
}

void square_minus(float *d_center, int n, int m, float *d_point, float *d_diff) {
    kernel_square_minus<<<m, n>>>(d_center, n, m, d_point, d_diff);
}

void set_uniform_value(float *d_m, int n, float min, float max) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(generator, rand());
    hiprandGenerateUniform(generator, d_m, n);
    kernel_uniform_scale <<<blocksPerGrid, threadsPerBlock>>>(d_m, n, min, max);
    hiprandDestroyGenerator(generator);
}

void set_uniform_value(float *d_m, int n, float epsilon) {
    set_uniform_value(d_m, n, 0, epsilon);
}

void shake(float *d_m, float *d_scale ,int n) {
    kernel_scale<<<(n - 1 + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(d_m, d_scale, n);
}